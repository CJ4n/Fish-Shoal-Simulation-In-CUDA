#include "hip/hip_runtime.h"
#include "boid.h"

Boid *init_boids()
{
    srand(0);
    Boid *boids;
    hipMallocManaged(&boids, sizeof(Boid));
    hipMallocManaged(&(boids->coord), sizeof(*(boids->coord)) * num_boids);
    hipMallocManaged(&(boids->velocity), sizeof(*(boids->velocity)) * num_boids);

    for (int boid = 0; boid < num_boids; ++boid)
    {
        float x = rand() % box_size;
        float y = rand() % box_size;
        float z = (is_3d == 1) ? rand() % box_size : 0;
        boids->coord[boid] = make_float3(x, y, z);

        float x_v = rand() % velocity_num_degres;
        float y_v = rand() % velocity_num_degres;
        float z_v = (is_3d == 1) ? rand() % velocity_num_degres : 0;
        float sum_v = x_v + y_v;
        if (sum_v > 0)
        {
            x_v = x_v / sum_v;
            y_v = y_v / sum_v;
        }
        if (rand() % 2 == 0)
        {
            x_v = -x_v;
        }

        if (rand() % 2 == 0)
        {
            y_v = -y_v;
        }
        if (rand() % 2 == 0)
        {
            z_v = -z_v;
        }
        boids->velocity[boid] = make_float3(x_v, y_v, z_v);
        // std::cout << "x:" << x << ", y: " << y << ", x_v: " << boids->velocity[boid].x << ", y_v: " << boids->velocity[boid].y << std::endl;
    }
    return boids;
}

__global__ void draw_boids(float3 *pos, Boid *boids, int num_boids)
{
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= num_boids)
    {
        return;
    }
    float x = boids->coord[gid].x;
    float y = boids->coord[gid].y;
    float z = boids->coord[gid].z;

    float x_v = boids->velocity[gid].x;
    float y_v = boids->velocity[gid].y;
    float z_v = boids->velocity[gid].z;

    float x_p1 = (x + x_v * size_boid) / (float)mesh_width * 2;
    float y_p1 = (y + y_v * size_boid) / (float)mesh_height * 2;
    float z_p1 = (z + z_v * size_boid) / (float)mesh_height * 2;

    float x_p2 = (x - y_v * size_boid / 2) / (float)mesh_width * 2;
    float y_p2 = (y + x_v * size_boid / 2) / (float)mesh_height * 2;
    float z_p2 = (z + z_v * size_boid / 2) / (float)mesh_height * 2;

    float x_p3 = (x + y_v * size_boid / 2) / (float)mesh_width * 2;
    float y_p3 = (y - x_v * size_boid / 2) / (float)mesh_height * 2;
    float z_p3 = (z + z_v * size_boid / 2) / (float)mesh_height * 2;

    pos[gid * 3] = make_float3((float)x_p1 - 1, (float)y_p1 - 1, (is_3d == 1) ? (float)z_p1 - 1 : 1.2);
    pos[gid * 3 + 1] = make_float3((float)x_p2 - 1, (float)y_p2 - 1, (is_3d == 1) ? (float)z_p2 - 1 : 1.2);
    pos[gid * 3 + 2] = make_float3((float)x_p3 - 1, (float)y_p3 - 1, (is_3d == 1) ? (float)z_p3 - 1 : 1.2);
}

// __device__ float dist(float x_1, float y_1, float x_2, float y_2)
// {
//     return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
// }

__global__ void update_boids_position(Boid *boids, float interaction_radius_2, float velocity, float factor_separation,
                                      float factor_alignment, float factor_cohesion, float factor_intertia)
{
    const int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (num_boids <= gid)
    {
        return;
    }

    DataType pos = boids->coord[gid];
    DataType vel = boids->velocity[gid];

    DataType pos_average = make_float3(0, 0, 0);
    DataType vel_average = make_float3(0, 0, 0);
    int count = 0;
    DataType new_velocity = vel;

    for (int idx = 0; idx < num_boids; ++idx)
    {
        DataType neighbour = boids->coord[idx];
        // if (dist(x, y, x_neighbour, y_neighbour) > interaction_radius_2)
        float d = (pos.x - neighbour.x) * (pos.x - neighbour.x) + (pos.y - neighbour.y) * (pos.y - neighbour.y) + (pos.z - neighbour.z) * (pos.z - neighbour.z);
        if (d > interaction_radius_2)
        {
            continue;
        }
        if (idx == gid)
        {
            continue;
        }
        pos_average.x += neighbour.x;
        pos_average.y += neighbour.y;
        if (is_3d)
            pos_average.z += neighbour.z;
        vel_average.x += boids->velocity[idx].x;
        vel_average.y += boids->velocity[idx].y;
        if (is_3d)
            vel_average.z += boids->velocity[idx].z;
        count++;
    }

    if (count > 0)
    {
        vel_average.x /= (float)count;
        vel_average.y /= (float)count;
        // if (is_3d)
        vel_average.z /= (float)count;
        pos_average.x /= (float)count;
        pos_average.y /= (float)count;
        // if (is_3d)
        pos_average.z /= (float)count;

        float sum_v_acumualte = std::abs(vel_average.x) + std::abs(vel_average.y) + std::abs(vel_average.z);
        float sum_acumulate = std::abs(pos_average.x) + std::abs(pos_average.y) + std::abs(pos_average.z);
        if (sum_v_acumualte > 0)
        {
            vel_average.x /= sum_v_acumualte;
            vel_average.y /= sum_v_acumualte;
            vel_average.z /= sum_v_acumualte;
        }
        DataType sep, coh, ali;

        // calculate separation force
        sep = make_float3(pos.x - pos_average.x, pos.y - pos_average.y, pos.z - pos_average.z);
        float sum_sep = std::abs(sep.x) + std::abs(sep.y) + std::abs(sep.z);
        if (sum_sep > 0)
        {
            sep.x /= sum_sep;
            sep.y /= sum_sep;
            sep.z /= sum_sep;
        }
        DataType separation_force = make_float3(factor_separation * sep.x, factor_separation * sep.y, factor_separation * sep.z);
        // calculate separation force

        // calculate alignment force
        ali = make_float3(vel_average.x - vel.x, vel_average.y - vel.y, vel_average.z - vel.z);
        float sum_ali = std::abs(ali.x) + std::abs(ali.y) + std::abs(ali.z);
        if (sum_ali > 0)
        {
            ali.x /= sum_ali;
            ali.y /= sum_ali;
            ali.z /= sum_ali;
        }
        DataType alignment_force = make_float3(factor_alignment * ali.x, factor_alignment * ali.y, factor_alignment * ali.z);
        // calculate alignment force

        // calculate cohision force
        coh = make_float3(pos_average.x - pos.x, pos_average.y - pos.y, pos_average.z - pos.z);
        float sum_coh = std::abs(coh.x) + std::abs(coh.y) + +std::abs(coh.z);
        if (sum_coh > 0)
        {
            coh.x /= sum_coh;
            coh.y /= sum_coh;
            coh.z /= sum_coh;
        }
        DataType cohesion_force = make_float3(factor_cohesion * coh.x, factor_cohesion * coh.y, factor_cohesion * coh.z);
        // calculate cohision force
        new_velocity = make_float3(factor_intertia * vel.x + separation_force.x + cohesion_force.x + alignment_force.x,
                                   factor_intertia * vel.y + separation_force.y + cohesion_force.y + alignment_force.y,
                                   factor_intertia * vel.z + separation_force.z + cohesion_force.z + alignment_force.z);

        float sum_vel = std::abs(new_velocity.x) + std::abs(new_velocity.y) + std::abs(new_velocity.z);
        if (sum_vel > 0)
        {
            new_velocity.x /= sum_vel;
            new_velocity.y /= sum_vel;
            new_velocity.z /= sum_vel;
        }
    }

    DataType new_pos = make_float3(pos.x + new_velocity.x * velocity, pos.y + new_velocity.y * velocity, pos.z + new_velocity.z * velocity);

    if (new_pos.x >= box_size || new_pos.x <= 0)
    {
        if (new_pos.x >= box_size)
        {
            new_pos.x = box_size;
        }
        else
        {
            new_pos.x = 0;
        }
        new_velocity.x = -new_velocity.x;
    }

    if (new_pos.y >= box_size || new_pos.y <= 0)
    {
        if (new_pos.y >= box_size)
        {
            new_pos.y = box_size;
        }
        else
        {
            new_pos.y = 0;
        }
        new_velocity.y = -new_velocity.y;
    }

    if (new_pos.z >= box_size || new_pos.z <= 0)
    {
        if (new_pos.z >= box_size)
        {
            new_pos.z = box_size;
        }
        else
        {
            new_pos.z = 0;
        }
        new_velocity.z = -new_velocity.z;
    }
    boids->velocity[gid] = new_velocity;
    boids->coord[gid] = new_pos;
}
