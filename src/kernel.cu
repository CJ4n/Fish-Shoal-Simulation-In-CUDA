#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include "kernel.h"
#include "constants.h"
#include <stdio.h>

// __global__ void simple_vbo_kernel(float3 *pos, int width, int height, float time)
// {
//     unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//     unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

//     // write output vertex
//     pos[y * width * 3 + x * 3] = make_float3(0, 0, 0);
//     pos[y * width * 3 + x * 3 + 1] = make_float3(0, 0, 0);
//     pos[y * width * 3 + x * 3 + 2] = make_float3(0, 0, 0);
// }

__global__ void draw_boids(float3 *pos, Boid *boids, int num_boids)
{
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= num_boids)
    {
        return;
    }
    MyType x = boids->coord[gid].x;
    MyType y = boids->coord[gid].y;

    MyType x_v = boids->velocity[gid].x;
    MyType y_v = boids->velocity[gid].y;

    MyType x_p1 = (x + x_v * size_boid) / (MyType)mesh_width;
    MyType y_p1 = (y + y_v * size_boid) / (MyType)mesh_height;

    MyType x_p2 = (x - y_v * size_boid / 2) / (MyType)mesh_width;
    MyType y_p2 = (y + x_v * size_boid / 2) / (MyType)mesh_height;

    MyType x_p3 = (x + y_v * size_boid / 2) / (MyType)mesh_width;
    MyType y_p3 = (y - x_v * size_boid / 2) / (MyType)mesh_height;

    pos[gid * 3] = make_float3((float)x_p1, (float)y_p1, 1.2);
    pos[gid * 3 + 1] = make_float3((float)x_p2, (float)y_p2, 1.2);
    pos[gid * 3 + 2] = make_float3((float)x_p3, (float)y_p3, 1.2);
}

__device__ MyType dist(MyType x_1, MyType y_1, MyType x_2, MyType y_2)
{
    return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

__global__ void update_boids_position(Boid *boids)
{
    const int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (num_boids <= gid)
    {
        return;
    }

    DataType pos = boids->coord[gid];
    DataType vel = boids->velocity[gid];

    DataType pos_acumulate = make_float2(0, 0);
    DataType vel_acumulate = make_float2(0, 0);
    int count = 0;
    DataType new_velocity = vel;

    for (int idx = 0; idx < num_boids; ++idx)
    {
        // continue;
        DataType neighbour = boids->coord[idx];
        // if (dist(x, y, x_neighbour, y_neighbour) > interaction_radius_2)
        float d = (pos.x - neighbour.x) * (pos.x - neighbour.x) + (pos.y - neighbour.y) * (pos.y - neighbour.y);
        if (d > interaction_radius_2)
        {
            continue;
        }
        if (idx == gid)
        {
            continue;
        }
        pos_acumulate.x += neighbour.x;
        pos_acumulate.y += neighbour.y;
        vel_acumulate.x += boids->velocity[idx].x;
        vel_acumulate.y += boids->velocity[idx].y;
        count++;
    }

    if (count > 0)
    {
        vel_acumulate.x /= (MyType)count;
        vel_acumulate.y /= (MyType)count;
        pos_acumulate.x /= (MyType)count;
        pos_acumulate.y /= (MyType)count;

        MyType sum_v_acumualte = std::abs(vel_acumulate.x) + std::abs(vel_acumulate.y);
        MyType sum_acumulate = std::abs(pos_acumulate.x) + std::abs(pos_acumulate.y);
        if (sum_v_acumualte > 0)
        {
            vel_acumulate.x /= sum_v_acumualte;
            vel_acumulate.y /= sum_v_acumualte;
        }
        DataType sep, coh, ali;
        sep = make_float2(pos.x - pos_acumulate.x, pos.y - pos_acumulate.y);
        MyType sum_sep = std::abs(sep.x) + std::abs(sep.y);
        if (sum_sep > 0)
        {
            sep.x /= sum_sep;
            sep.y /= sum_sep;
        }

        coh = make_float2(pos_acumulate.x - pos.x, pos_acumulate.y - pos.y);
        MyType sum_coh = std::abs(coh.x) + std::abs(coh.y);
        if (sum_coh > 0)
        {
            coh.x /= sum_coh;
            coh.y /= sum_coh;
        }

        ali = make_float2(vel_acumulate.x - vel.x, vel_acumulate.y - vel.y);
        MyType sum_ali = std::abs(ali.x) + std::abs(ali.y);
        if (sum_ali > 0)
        {
            ali.x /= sum_ali;
            ali.y /= sum_ali;
        }
        DataType separation_force = make_float2(factor_separation * sep.x, factor_separation * sep.y);
        DataType cohesion_force = make_float2(factor_cohesion * coh.x, factor_cohesion * coh.y);
        DataType alignment_force = make_float2(factor_alignment * ali.x, factor_alignment * ali.y);
        new_velocity = make_float2(factor_intertia * vel.x + separation_force.x + cohesion_force.x + alignment_force.x,
                                   factor_intertia * vel.y + separation_force.y + cohesion_force.y + alignment_force.y);

        MyType sum_vel = std::abs(new_velocity.x) + std::abs(new_velocity.y);
        if (sum_vel > 0)
        {
            new_velocity.x /= sum_vel;
            new_velocity.y /= sum_vel;
        }
    }
    DataType new_pos = make_float2(pos.x + new_velocity.x * velocity, pos.y + new_velocity.y * velocity);

    if (new_pos.x >= box_size || new_pos.x <= 0)
    {
        new_velocity.x = -new_velocity.x;
    }

    if (new_pos.y >= box_size || new_pos.y <= 0)
    {
        new_velocity.y = -new_velocity.y;
    }
    boids->velocity[gid] = new_velocity;
    boids->coord[gid] = new_pos;
}

Boid *init_boids()
{
    srand(0);
    Boid *boids;
    hipMallocManaged(&boids, sizeof(Boid));

    hipMallocManaged(&(boids->coord), sizeof(*(boids->coord)) * num_boids);
    hipMallocManaged(&(boids->velocity), sizeof(*(boids->velocity)) * num_boids);

    for (int boid = 0; boid < num_boids; ++boid)
    {
        MyType x = rand() % box_size;
        MyType y = rand() % box_size;
        boids->coord[boid] = make_float2(x, y);

        MyType x_v = rand() % velocity_num_degres;
        MyType y_v = rand() % velocity_num_degres;
        MyType sum_v = x_v + y_v;
        if (sum_v > 0)
        {
            x_v = x_v / sum_v;
            y_v = y_v / sum_v;
        }
        if (rand() % 2 == 0)
        {
            x_v = -x_v;
        }

        if (rand() % 2 == 0)
        {
            y_v = -y_v;
        }
        boids->velocity[boid] = make_float2(x_v, y_v);

        // std::cout << "x:" << x << ", y: " << y << ", x_v: " << boids->velocity[boid].x << ", y_v: " << boids->velocity[boid].y << std::endl;
    }
    return boids;
}
void launch_kernel(float3 *pos, Boid *boids)
{

    const int num_threads = std::min(1024, num_boids);
    const int num_blocks = std::ceil((float)num_boids / (float)num_threads);
    update_boids_position<<<num_blocks, num_threads>>>(boids);
    draw_boids<<<num_blocks, num_threads>>>(pos, boids, num_boids);
    // for (int boid = 0; boid <num_boids; ++boid)
    // {
    //     std::cout << "x:" << boids->coord[boid].x << ", y: " << boids->coord[boid].y << ", x_v: " << boids->velocity[boid].x << ", y_v: " << boids->velocity[boid].y << std::endl;
    // }
}